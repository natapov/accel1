#include "hip/hip_runtime.h"
#include "ex1.h"
using namespace std;

const int img_size = SIZE * SIZE * CHANNELS;
__device__ void prefixSum(int arr[], int size, int tid, int threads) {
    int increment;
    const auto is_active = tid < size;
    for (int stride = 1; stride<size; stride*=2) {
        if (tid >= stride && is_active) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && is_active) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

__device__ void argmin(int arr[], int len, int tid, int threads) {
    int halfLen = len / 2;
    bool firstIteration = true;
    int prevHalfLength = 0;
    while (halfLen > 0) {
        if(tid < halfLen){
            if(arr[tid] == arr[tid + halfLen]){ //a corenr case
                int lhsIdx = tid;
                int rhdIdx = tid + halfLen;
                int lhsOriginalIdx = firstIteration ? lhsIdx : arr[prevHalfLength + lhsIdx];
                int rhsOriginalIdx = firstIteration ? rhdIdx : arr[prevHalfLength + rhdIdx];
                arr[tid + halfLen] = lhsOriginalIdx < rhsOriginalIdx ? lhsOriginalIdx : rhsOriginalIdx;
            }
            else { //the common case
                bool isLhsSmaller = (arr[tid] < arr[tid + halfLen]);
                int idxOfSmaller = isLhsSmaller * tid + (!isLhsSmaller) * (tid + halfLen);
                int smallerValue = arr[idxOfSmaller];
                int origIdxOfSmaller = firstIteration * idxOfSmaller + (!firstIteration) * arr[prevHalfLength + idxOfSmaller];
                arr[tid] = smallerValue;
                arr[tid + halfLen] = origIdxOfSmaller;
            }
        }
        __syncthreads();
        firstIteration = false;
        prevHalfLength = halfLen;
        halfLen /= 2;
    }
}

__device__ void zero_array(int* histograms, int size=CHANNELS*LEVELS) {
    auto hist_flat = (int*) histograms;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;
    for(int i = tid; i < size; i+=threads) {
        hist_flat[i] = 0;
    }
}

__device__ void colorHist(uchar img[][CHANNELS], int histograms[][LEVELS]) {
    const int pic_size = SIZE * SIZE;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;

    for (int i = tid; i < 3*pic_size; i+=threads) {
        const int color = i%3;
        const int pixel = i/3;
        assert(pixel < pic_size);
        atomicAdd(&histograms[color][img[pixel][color]], 1);
    }
}

__device__ void performMapping(int maps[][LEVELS], uchar targetImg[][CHANNELS], uchar resultImg[][CHANNELS]){
    int pixels = SIZE * SIZE;
    const int tid = threadIdx.x;
    const int threads = blockDim.x;
    for (int i = tid; i < pixels; i+= threads) {
        uchar *inRgbPixel = targetImg[i];
        uchar *outRgbPixel = resultImg[i];
        for (int j = 0; j < CHANNELS; j++){
            int *mapChannel = maps[j];
            outRgbPixel[j] = mapChannel[inRgbPixel[j]];
        }
    }    
}


__global__
void process_image_kernel(uchar *targets, uchar *refrences, uchar *results) {
    int tid = threadIdx.x;;
    int threads = blockDim.x;
    int bid = blockIdx.x;
    __shared__ int deleta_cdf_row[LEVELS];
    __shared__ int map_cdf[CHANNELS][LEVELS];
    __shared__ int histogramsShared_target[CHANNELS][LEVELS];
    __shared__ int histogramsShared_refrence[CHANNELS][LEVELS];
    zero_array((int*)histogramsShared_target,   CHANNELS * LEVELS);
    zero_array((int*)histogramsShared_refrence, CHANNELS * LEVELS);
    zero_array((int*)map_cdf,                   CHANNELS * LEVELS);
    zero_array((int*)deleta_cdf_row,            LEVELS);

    auto target   = (uchar(*)[CHANNELS]) &targets[  bid * img_size];
    auto refrence = (uchar(*)[CHANNELS]) &refrences[bid * img_size];
    auto result   = (uchar(*)[CHANNELS]) &results[  bid * img_size];

    colorHist(target, histogramsShared_target);
    colorHist(refrence, histogramsShared_refrence);
    __syncthreads();

    for(int c=0; c < CHANNELS; c++)
    {   
        prefixSum(histogramsShared_target[c],LEVELS, threadIdx.x, blockDim.x);
        prefixSum(histogramsShared_refrence[c], LEVELS, threadIdx.x, blockDim.x);
        __syncthreads();

        for (int i = 0; i < LEVELS; i+=1) {
            for (int j = tid; j < LEVELS; j+=threads) {
                deleta_cdf_row[j] = abs(histogramsShared_target[c][i]-histogramsShared_refrence[c][j]);
            }
            __syncthreads();
            argmin(deleta_cdf_row, LEVELS, threadIdx.x, blockDim.x);
            __syncthreads();

            map_cdf[c][i] = deleta_cdf_row[1];

            __syncthreads();
        }
        __syncthreads();
    }          

    //Preform Map
    performMapping(map_cdf, target, result); 
    __syncthreads(); 
}


/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    uchar *target_single   = nullptr;
    uchar *refrence_single = nullptr;
    uchar *result_single   = nullptr;
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context* task_serial_init()
{
    auto context = new task_serial_context;
    //allocate GPU memory for a single input image and a single output image
    CUDA_CHECK( hipMalloc((void**)&(context->target_single),   img_size) ); 
    CUDA_CHECK( hipMalloc((void**)&(context->refrence_single), img_size) ); 
    CUDA_CHECK( hipMalloc((void**)&(context->result_single),   img_size) ); 
    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_target, uchar *images_refrence, uchar *images_result)
{
    for(int i = 0; i < N_IMAGES; i++) {
        CUDA_CHECK( hipMemcpy(context->target_single,   images_target,   img_size, hipMemcpyHostToDevice) );
        CUDA_CHECK( hipMemcpy(context->refrence_single, images_refrence, img_size, hipMemcpyHostToDevice) );
        process_image_kernel<<<1,256>>>(context->target_single, context->refrence_single, context->result_single);
        CUDA_CHECK( hipMemcpy(images_result, context->result_single, img_size, hipMemcpyDeviceToHost) );
        images_target   += img_size;
        images_refrence += img_size;
        images_result   += img_size;
    }
}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    hipFree(context->refrence_single);
    hipFree(context->target_single);
    hipFree(context->result_single);
    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    // define bulk-GPU memory buffers
    uchar *target_single   = nullptr;
    uchar *refrence_single = nullptr;
    uchar *result_single   = nullptr;
};

/* Allocate GPU memory for all the input and output images.
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    // allocate GPU memory for all input images and all output images
    CUDA_CHECK( hipMalloc((void**)&(context->target_single),  N_IMAGES * img_size) );
    CUDA_CHECK( hipMalloc((void**)&(context->refrence_single),N_IMAGES * img_size) );
    CUDA_CHECK( hipMalloc((void**)&(context->result_single),  N_IMAGES * img_size) );
    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_target, uchar *images_refrence, uchar *images_result)
{
    // copy all input images from images_in to the GPU memory you allocated
    // invoke a kernel with N_IMAGES threadblocks, each working on a different image
    // copy output images from GPU memory to images_out
    CUDA_CHECK( hipMemcpy(context->target_single,   images_target,   N_IMAGES * img_size, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(context->refrence_single, images_refrence, N_IMAGES * img_size, hipMemcpyHostToDevice) );
    process_image_kernel<<<N_IMAGES, 256>>>(context->target_single, context->refrence_single, context->result_single);
    CUDA_CHECK( hipMemcpy(images_result, context->result_single, N_IMAGES * img_size, hipMemcpyDeviceToHost) );
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    // free resources allocated in gpu_bulk_init
    hipFree(context->refrence_single);
    hipFree(context->target_single);
    hipFree(context->result_single);
    free(context);
}


/********************************************************
**  the following waappers are needed for unit testing.
********************************************************/

__global__ void argminWrapper(int arr[], int size){
    argmin(arr, size, threadIdx.x, blockDim.x);
}

__global__ void colorHistWrapper(uchar img[][CHANNELS], int histograms[][LEVELS]){
    __shared__ int histogramsShared[CHANNELS][LEVELS];   
    zero_array((int*)histogramsShared);
    __syncthreads();

    int tid = threadIdx.x;
    int threads = blockDim.x;
    colorHist(img, histogramsShared);
    __syncthreads();


    for(int i = tid; i < CHANNELS * LEVELS; i+=threads){
        ((int*)histograms)[i] = ((int*)histogramsShared)[i];
    }
    __syncthreads();

}

__global__ void prefixSumWrapper(int arr[], int size){
    __shared__ int arrShared[LEVELS];

    int tid = threadIdx.x;
    int threads = blockDim.x;

    for(int i=tid; i<size; i+=threads){
        arrShared[i] = arr[i];
    }

    __syncthreads();

    prefixSum(arrShared, size, threadIdx.x, blockDim.x);
    
    __syncthreads();

    for(int i=tid; i<size; i+=threads){
        arr[i] = arrShared[i];
    }

    __syncthreads();
}

__global__ void performMappingWrapper(int maps[][LEVELS], uchar targetImg[][CHANNELS], uchar resultImg[][CHANNELS]){
    __syncthreads();
    performMapping(maps, targetImg, resultImg);
    __syncthreads();
}
