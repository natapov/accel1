#include "hip/hip_runtime.h"
///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////

#include "ex1.h"

void colorHist(uchar img[][CHANNELS], int pixelCount, int histograms[][LEVELS]){
    memset(histograms, 0, sizeof(int) * CHANNELS * LEVELS);
    for (int i = 0; i < pixelCount; i++) {
        uchar *rgbPixel = img[i];
        for (int j = 0; j < CHANNELS; j++){
            int *channelHist = histograms[j];
            channelHist[rgbPixel[j]] += 1;
        }
    }
}


void prefixSum(int arr[], int size, int res[]){
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += arr[i];
        res[i] = sum;
    }
}

void imgHistCdf(uchar img[][CHANNELS], int pixelCount, uchar res[][LEVELS]){
    int histograms[CHANNELS][LEVELS];
    int tmp[CHANNELS][LEVELS];
    colorHist(img, pixelCount, histograms);
    for (int j = 0; j < CHANNELS; j++){
        int *channelHist = histograms[j];
        prefixSum(channelHist, LEVELS, tmp[j]);
    }
    for(int j = 0; j < CHANNELS; j++) {
        const int max = tmp[j][LEVELS -1];
        for(int i = 0; i < LEVELS; i++) {
            res[j][i] = (uchar) ((tmp[j][i]*256)/max);
        } 
    }
}

int argmin(int arr[], int size){
    int argmin = -1;
    int min = INT_MAX;
    for(int i = 0; i < size; i++){
        if (arr[i] < min){
            min = arr[i];
            argmin = i;
        }
    }
    return argmin;
}

void calculateMap(uchar targetCdf[], uchar refrenceCdf[],  uchar map[]){
    int diff[LEVELS][LEVELS];
    for(int i_ref = 0; i_ref < LEVELS; i_ref++){
        for(int i_tar = 0; i_tar < LEVELS; i_tar++){
            diff[i_tar][i_ref] = (int) abs(refrenceCdf[i_ref] - targetCdf[i_tar]);
        }
    }

    for(int row = 0; row < LEVELS; row++){
        map[row] = (uchar) argmin(diff[row], LEVELS);
    }
}

void performMapping(uchar maps[][LEVELS], uchar targetImg[][CHANNELS], uchar resultImg[][CHANNELS], int width, int height){
    int pixels = width * height;
    for (int i = 0; i < pixels; i++) {
        uchar *inRgbPixel = targetImg[i];
        uchar *outRgbPixel = resultImg[i];
        for (int j = 0; j < CHANNELS; j++){
            uchar *mapChannel = maps[j];
            outRgbPixel[j] = mapChannel[inRgbPixel[j]];
        }
    }
}

void cpu_process(uchar targetImg[][CHANNELS], uchar refrenceImg[][CHANNELS],  uchar outputImg[][CHANNELS], int width, int height) {
    uchar targetCdf[CHANNELS][LEVELS];
    uchar refrenceCdf[CHANNELS][LEVELS];
    uchar maps[CHANNELS][LEVELS];
    int pixelCount = width * height;
    imgHistCdf(targetImg, pixelCount, targetCdf);
    imgHistCdf(refrenceImg, pixelCount, refrenceCdf);
    
    for (int i = 0; i < CHANNELS; i++){
        uchar *refrencetChannelCdf = refrenceCdf[i];
        uchar *targetChannelCdf = targetCdf[i];
        uchar *mapChannel = maps[i];
        calculateMap(targetChannelCdf, refrencetChannelCdf, mapChannel);
    }

    performMapping(maps, targetImg, outputImg, width, height);
}


